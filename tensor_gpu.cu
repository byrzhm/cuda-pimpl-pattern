#include "hip/hip_runtime.h"
#include "tensor_impl.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

__global__ void fill_kernel(float *data, int n, float value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    data[idx] = value;
}

__global__ void add_kernel(float *a, const float *b, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    a[idx] += b[idx];
}

struct TensorGPUImpl : TensorImplBase {
  float *data;
  int size;

  TensorGPUImpl(int n) : size(n) { hipMalloc(&data, n * sizeof(float)); }

  ~TensorGPUImpl() override { hipFree(data); }

  void fill(float value) override {
    int threads = 128;
    int blocks = (size + threads - 1) / threads;
    fill_kernel<<<blocks, threads>>>(data, size, value);
    hipDeviceSynchronize();
  }

  void add(const TensorImplBase &other) override {
    const auto &rhs = dynamic_cast<const TensorGPUImpl &>(other);
    int threads = 128;
    int blocks = (size + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(data, rhs.data, size);
    hipDeviceSynchronize();
  }

  void print() const override {
    std::vector<float> host(size);
    hipMemcpy(host.data(), data, size * sizeof(float), hipMemcpyDeviceToHost);
    for (auto v : host)
      std::cout << v << " ";
    std::cout << "\n";
  }
};

std::unique_ptr<TensorImplBase> make_gpu_tensor(int size) {
  return std::make_unique<TensorGPUImpl>(size);
}

